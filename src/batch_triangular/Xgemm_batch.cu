/**
 * @copyright (c) 2012- King Abdullah University of Science and
 *                      Technology (KAUST). All rights reserved.
 **/


/**
 * @file src/batch_triangular/Xgemm_batch.cu

 * KBLAS is a high performance CUDA library for subset of BLAS
 *    and LAPACK routines optimized for NVIDIA GPUs.
 * KBLAS is provided by KAUST.
 *
 * @version 4.0.0
 * @author Ali Charara
 * @date 2020-12-10
 **/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hipblas.h"
#include "kblas.h"
#include "kblas_operators.h"
#include <typeinfo>


#include "kblas_struct.h"
#include "kblas_prec_def.h"

#include "kblas_common.h"
#include "workspace_queries.ch"
#include "Xhelper_funcs.ch"
#include "Xgemm_batch_core.cuh"

//=================================================================================
//Non-Strided form

/**
 * Workspace needed: device pointers
 *
 * @param[in] A_row_off row offset to sub-matrix of all A's
 * @param[in] A_col_off column offset to sub-matrix of all A's
 * @param[in] B_row_off row offset to sub-matrix of all B's
 * @param[in] B_col_off column offset to sub-matrix of all B's
 * @param[in] C_row_off row offset to sub-matrix of all C's
 * @param[in] C_col_off column offset to sub-matrix of all C's
 * @see kblasSgemm_batch() for details about rest of params.
 * A, B, C: host pointer to array of device pointers to device buffers
 */
int kblas_gemm_batch( kblasHandle_t handle,
                      char transA, char transB,
                      const int m, const int n, const int k,
                      const TYPE alpha,
                      const TYPE** A, int A_row_off, int A_col_off, int lda,
                      const TYPE** B, int B_row_off, int B_col_off, int ldb,
                      const TYPE beta,
                            TYPE** C, int C_row_off, int C_col_off, int ldc,
                      int batchCount)
{
  return Xgemm_batch_uniform_core(handle,
                                  transA, transB,
                                  m, n, k,
                                  alpha,
                                  A, A_row_off, A_col_off, lda,
                                  B, B_row_off, B_col_off, ldb,
                                  beta,
                                  C, C_row_off, C_col_off, ldc,
                                  batchCount);
}
//--------------------------------------------
int Xgemm_batch(kblasHandle_t handle,
                char transA, char transB,
                int m, int n, int k,
                TYPE alpha,
                TYPE** A, int A_row_off, int A_col_off, int lda, long strideA,
                TYPE** B, int B_row_off, int B_col_off, int ldb, long strideB,
                TYPE beta,
                TYPE** C, int C_row_off, int C_col_off, int ldc, long strideC,
                int batchCount)
{
  (void)strideA;
  (void)strideB;
  (void)strideC;
  return Xgemm_batch_uniform_core(handle,
                                  transA, transB,
                                  m, n, k,
                                  alpha,
                                  (const TYPE**)A, A_row_off, A_col_off, lda,
                                  (const TYPE**)B, B_row_off, B_col_off, ldb,
                                  beta,
                                        (TYPE**)C, C_row_off, C_col_off, ldc,
                                  batchCount);
}

//--------------------------------------------
// Workspace needed: none
int Xgemm_batch(kblasHandle_t handle,
                char transA, char transB,
                int m, int n, int k,
                TYPE alpha,
                TYPE** A_array, int lda, long strideA,
                TYPE** B_array, int ldb, long strideB,
                TYPE beta,
                TYPE** C_array, int ldc, long strideC,
                int batchCount)
{
  (void)strideA;
  (void)strideB;
  (void)strideC;
  return Xgemm_batch_uniform_core(handle,
                                  transA, transB,
                                  m, n, k,
                                  alpha,
                                  (const TYPE**)A_array, 0, 0, lda,
                                  (const TYPE**)B_array, 0, 0, ldb,
                                  beta,
                                        (TYPE**)C_array, 0, 0, ldc,
                                  batchCount);
}

//--------------------------------------------
// Workspace needed: none
int kblas_gemm_batch( kblasHandle_t handle,
                      char transA, char transB,
                      const int m, const int n, const int k,
                      const TYPE alpha,
                      const TYPE** A_array, int lda,
                      const TYPE** B_array, int ldb,
                      const TYPE beta,
                            TYPE** C_array, int ldc,
                      int batchCount)
{
  return Xgemm_batch_uniform_core(handle,
                                  transA, transB,
                                  m, n, k,
                                  alpha,
                                  A_array, 0, 0, lda,
                                  B_array, 0, 0, ldb,
                                  beta,
                                  C_array, 0, 0, ldc,
                                  batchCount);
}

//--------------------------------------------
// Workspace needed: none
extern "C"
int kblasXgemm_batch( kblasHandle_t handle,
                      char transA, char transB,
                      const int m, const int n, const int k,
                      const TYPE alpha,
                      const TYPE** A, int lda,
                      const TYPE** B, int ldb,
                      const TYPE beta,
                            TYPE** C, int ldc,
                      int batchCount)
{
  return Xgemm_batch_uniform_core(handle,
                                  transA, transB,
                                  m, n, k,
                                  alpha,
                                  A, 0, 0, lda,
                                  B, 0, 0, ldb,
                                  beta,
                                  C, 0, 0, ldc,
                                  batchCount);
}

//--------------------------------------------
int Xgemm_batch(kblasHandle_t handle,
                char transA, char transB,
                int* m, int* n, int* k,
                int max_m, int max_n, int max_k,
                const TYPE alpha,
                const TYPE** A, int A_row_off, int A_col_off, int* lda,
                const TYPE** B, int B_row_off, int B_col_off, int* ldb,
                const TYPE beta,
                      TYPE** C, int C_row_off, int C_col_off, int* ldc,
                int batchCount )
{
  return Xgemm_batch_nonuniform_core<TYPE>(
                                    handle,
                                    transA, transB,
                                    m, n, k,
                                    alpha,
                                    A, A_row_off, A_col_off, lda,
                                    B, B_row_off, B_col_off, ldb,
                                    beta,
                                    C, C_row_off, C_col_off, ldc,
                                    max_m, max_n, max_k,
                                    batchCount );
}

//--------------------------------------------
int kblas_gemm_batch( kblasHandle_t handle,
                      char transA, char transB,
                      int* m, int* n, int* k,
                      int max_m, int max_n, int max_k,
                      const TYPE alpha,
                      const TYPE** A, int* lda,
                      const TYPE** B, int* ldb,
                      const TYPE beta,
                            TYPE** C, int* ldc,
                      int batchCount )
{
  return Xgemm_batch( handle,
                      transA, transB,
                      m, n, k,
                      max_m, max_n, max_k,
                      alpha,
                      A, 0, 0, lda,
                      B, 0, 0, ldb,
                      beta,
                      C, 0, 0, ldc,
                      batchCount );
}

// //--------------------------------------------
// int kblas_gemm_batch( kblasHandle_t handle,
//                       char transA, char transB,
//                       int* m, int* n, int* k,
//                       const TYPE alpha,
//                       const TYPE** A, int A_row_off, int A_col_off, int* lda,
//                       const TYPE** B, int B_row_off, int B_col_off, int* ldb,
//                       const TYPE beta,
//                             TYPE** C, int C_row_off, int C_col_off, int* ldc,
//                       int batchCount )
// {
//   return Xgemm_batch_nonuniform_core<TYPE>(
//                                     handle,
//                                     transA, transB,
//                                     m, n, k,
//                                     alpha,
//                                     A, A_row_off, A_col_off, lda,
//                                     B, B_row_off, B_col_off, ldb,
//                                     beta,
//                                     C, C_row_off, C_col_off, ldc,
//                                     batchCount );
// }

//--------------------------------------------
int kblas_gemm_batch( kblasHandle_t handle,
                      char transA, char transB,
                      int* m, int* n, int* k,
                      const TYPE alpha,
                      const TYPE** A, int* lda,
                      const TYPE** B, int* ldb,
                      const TYPE beta,
                            TYPE** C, int* ldc,
                      int batchCount )
{
  return Xgemm_batch( handle,
                      transA, transB,
                      m, n, k,
                      0, 0, 0,
                      alpha,
                      A, 0, 0, lda,
                      B, 0, 0, ldb,
                      beta,
                      C, 0, 0, ldc,
                      batchCount );
}
//==============================================================================================
//Strided form

//TODO IMPORTANT: stride should be long long int since it is a memory address measure


/**
 * Uniform-size batch strided GEMM wrapper
 *
 * Workspace needed= ( __CUDACC_VER_MAJOR__ < 8 ) ? device pointers : none
 * A, B, C: host pointers to device buffers
 */
int Xgemm_batch(kblasHandle_t handle,
                char transA, char transB,
                int m, int n, int k,
                TYPE alpha,
                TYPE* A, int lda, long strideA,
                TYPE* B, int ldb, long strideB,
                TYPE beta,
                TYPE* C, int ldc, long strideC,
                int batchCount)
{
  return Xgemm_batch_uniform_core(handle,
                                  transA, transB,
                                  m, n, k,
                                  alpha,
                                  (const TYPE*)A, 0, 0, lda, strideA,
                                  (const TYPE*)B, 0, 0, ldb, strideB,
                                  beta,
                                        (TYPE*)C, 0, 0, ldc, strideC,
                                  batchCount);
}
//--------------------------------------------
int kblas_gemm_batch( kblasHandle_t handle,
                      char transA, char transB,
                      const int m, const int n, const int k,
                      const TYPE alpha,
                      const TYPE* A, int lda, long strideA,
                      const TYPE* B, int ldb, long strideB,
                      const TYPE beta,
                            TYPE* C, int ldc, long strideC,
                      int batchCount)
{
  return Xgemm_batch_uniform_core(handle,
                                  transA, transB,
                                  m, n, k,
                                  alpha,
                                  A, 0, 0, lda, strideA,
                                  B, 0, 0, ldb, strideB,
                                  beta,
                                  C, 0, 0, ldc, strideC,
                                  batchCount);
}

//--------------------------------------------
int Xgemm_batch(kblasHandle_t handle,
                char transA, char transB,
                int m, int n, int k,
                TYPE alpha,
                TYPE* A, int A_row_off, int A_col_off, int lda, long strideA,
                TYPE* B, int B_row_off, int B_col_off, int ldb, long strideB,
                TYPE beta,
                TYPE* C, int C_row_off, int C_col_off, int ldc, long strideC,
                int batchCount)
{
  return Xgemm_batch_uniform_core(handle,
                                  transA, transB,
                                  m, n, k,
                                  alpha,
                                  A, A_row_off, A_col_off, lda, strideA,
                                  B, B_row_off, B_col_off, ldb, strideB,
                                  beta,
                                  C, C_row_off, C_col_off, ldc, strideC,
                                  batchCount);
}

//--------------------------------------------
// A, B, C: host pointers to device buffers
extern "C"
int kblasXgemm_batch_strided( kblasHandle_t handle,
                              char transA, char transB,
                              const int m, const int n, const int k,
                              const TYPE alpha,
                              const TYPE* A, int lda, long strideA,
                              const TYPE* B, int ldb, long strideB,
                              const TYPE beta,
                                    TYPE* C, int ldc, long strideC,
                              int batchCount)
{
  return Xgemm_batch_uniform_core(handle,
                                  transA, transB,
                                  m, n, k,
                                  alpha,
                                  A, 0, 0, lda, strideA,
                                  B, 0, 0, ldb, strideB,
                                  beta,
                                  C, 0, 0, ldc, strideC,
                                  batchCount);
}
