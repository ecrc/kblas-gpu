/**
 * @copyright (c) 2012- King Abdullah University of Science and
 *                      Technology (KAUST). All rights reserved.
 **/


/**
 * @file src/batch_triangular/Xpotri_batch.cu

 * KBLAS is a high performance CUDA library for subset of BLAS
 *    and LAPACK routines optimized for NVIDIA GPUs.
 * KBLAS is provided by KAUST.
 *
 * @version 3.0.0
 * @author Ali Charara
 * @date 2018-11-14
 **/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hipblas.h"
#include <typeinfo>

#include "kblas.h"
#include "kblas_struct.h"
#include "kblas_operators.h"
#include "kblas_defs.h"
#include "kblas_common.h"
#include "workspace_queries.ch"

//==============================================================================================
#include "Xblas_core.ch"
#include "Xhelper_funcs.ch"
#include "Xpotri_batch_drivers.cuh"

//==============================================================================================
//Non-Strided form

// workspace needed: device pointers
// A: host pointer to device buffer
int Xpotri_batch_offset(kblasHandle_t handle,
                        char uplo,
                        const int n,
                        TYPE** A, int A_row_off, int A_col_off, int lda,
                        int batchCount,
                        int *info_array)
{
  KBlasWorkspaceState ws_needed;
  potri_batch_wsquery_core<false>( n, batchCount, (kblasWorkspaceState_t)&ws_needed);

  if( !ws_needed.isSufficient( &(handle->work_space.allocated_ws_state) ) ){
    return KBLAS_InsufficientWorkspace;
  }

  return Xpotri_batch_core<TYPE, TYPE**, false>(
                          handle,
                          uplo, n,
                          (TYPE**)A, A_row_off, A_col_off, lda, (long)0,
                          batchCount,
                          info_array);
}

// workspace needed: device pointers
// A: host pointer to device buffer
int kblas_potri_batch(kblasHandle_t handle,
                      char uplo,
                      const int n,
                      TYPE** A, int lda,
                      int batchCount,
                      int *info_array)
{
  return Xpotri_batch_offset( handle,
                              uplo, n,
                              A, 0, 0, lda,
                              batchCount,
                              info_array);
}


// workspace needed: device pointers
// A: host pointer to device buffer
extern "C"
int kblasXpotri_batch(kblasHandle_t handle,
                      char uplo,
                      const int n,
                      TYPE** A, int lda,
                      int batchCount,
                      int *info_array)
{
  return Xpotri_batch_offset( handle,
                              uplo, n,
                              A, 0, 0, lda,
                              batchCount,
                              info_array);
}


//==============================================================================================
//Strided form
// template<>

// workspace needed: device pointers
// A: host pointer to device buffer
int Xpotri_batch_offset(kblasHandle_t handle,
                        char uplo,
                        const int n,
                        TYPE* A, int A_row_off, int A_col_off, int lda, long strideA,
                        int batchCount,
                        int *info_array)
{
  KBlasWorkspaceState ws_needed;
  potri_batch_wsquery_core<true>( n, batchCount, (kblasWorkspaceState_t)&ws_needed);

  if( !ws_needed.isSufficient( &(handle->work_space.allocated_ws_state) ) ){
    return KBLAS_InsufficientWorkspace;
  }

  return Xpotri_batch_core<TYPE, TYPE*, true>(
                          handle,
                          uplo, n,
                          (TYPE*)A, A_row_off, A_col_off, lda, strideA,
                          batchCount,
                          info_array);
}

// workspace needed: device pointers
// A: host pointer to device buffer
int kblas_potri_batch(kblasHandle_t handle,
                      char uplo,
                      const int n,
                      TYPE* A, int lda, long strideA,
                      int batchCount,
                      int *info_array)
{
  return Xpotri_batch_offset( handle,
                              uplo, n,
                              A, 0, 0, lda, strideA,
                              batchCount,
                              info_array);
}

// workspace needed: device pointers
// A: host pointer to device buffer
extern "C"
int kblasXpotri_batch_strided(kblasHandle_t handle,
                              char uplo,
                              const int n,
                              TYPE* A, int lda, long strideA,
                              int batchCount,
                              int *info_array)
{
  return Xpotri_batch_offset( handle,
                              uplo, n,
                              A, 0, 0, lda, strideA,
                              batchCount,
                              info_array);
}
