/**
 * @copyright (c) 2012- King Abdullah University of Science and
 *                      Technology (KAUST). All rights reserved.
 **/


/**
 * @file src/blas_l2/chemv.cu

 * KBLAS is a high performance CUDA library for subset of BLAS
 *    and LAPACK routines optimized for NVIDIA GPUs.
 * KBLAS is provided by KAUST.
 *
 * @version 4.0.0
 * @author Ahmad Abdelfattah
 * @date 2020-12-10
 **/

#include "syhemv_core.cuh"

#if(TARGET_SM >= 30)

#define chemv_upper_bs	(32)
#define chemv_upper_ty	(2)
#define chemv_upper_by	(2)

#define chemv_lower_bs	(32)
#define chemv_lower_ty	(2)
#define chemv_lower_by	(2)

#else

#define chemv_upper_bs	(64)
#define chemv_upper_ty	(8)
#define chemv_upper_by	(2)

#define chemv_lower_bs	(32)
#define chemv_lower_ty	(4)
#define chemv_lower_by	(2)

#endif


int kblas_chemv_driver( char uplo,
						int m, hipFloatComplex alpha, hipFloatComplex *dA, int lda,
						hipFloatComplex *dX, int incx,
						hipFloatComplex  beta, hipFloatComplex *dY, int incy,
						hipStream_t stream)
{
	// handle the case when incx and/or incy is -ve
	if(incx < 0) dX -= (m-1) * incx;
	if(incy < 0) dY -= (m-1) * incy;

	if(uplo == 'U' || uplo == 'u')
	{
		/** configuration params **/
		/**
		* If you change the configuration parameters,
		* you must revise the case statement of the upper case
		* to make sure it covers all the possible cases
		**/
		const int chemv_bs = chemv_upper_bs;
		const int thread_x = chemv_bs;
		const int thread_y = chemv_upper_ty;
		const int elements_per_thread = (chemv_bs/(2*thread_y)) ;
		/** end configuration params **/

		int mod = m % chemv_bs;
		int blocks = m / chemv_bs + (mod != 0);
		dim3 dimBlock(thread_x, thread_y);
		dim3 dimGrid(blocks,1);
		dim3 dimGrid_(blocks, chemv_upper_by);

		if(mod == 0)
		{
		  syhemvu_special_d<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy);
		  syhemvu_special_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid_, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy);
		}
		else
		{
			syhemvu_generic_d<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod);
			/**
			* The upper case kernel for irregular dimensions has an extra template parameter.
			* This parameter must be among the values listed in the switch-case statement below.
			* The possible values are in the range 0 - (elements_per_thread-1)
			* Make sure these values are updated whenever you change the configuration parameters.
			**/
			const int irregular_part = mod % elements_per_thread;
			switch(irregular_part)
			{
				case  0: syhemvu_generic_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread,  0><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod); break;
				case  1: syhemvu_generic_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread,  1><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod); break;
				case  2: syhemvu_generic_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread,  2><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod); break;
				case  3: syhemvu_generic_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread,  3><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod); break;
				case  4: syhemvu_generic_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread,  4><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod); break;
				case  5: syhemvu_generic_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread,  5><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod); break;
				case  6: syhemvu_generic_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread,  6><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod); break;
				case  7: syhemvu_generic_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread,  7><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod); break;
				case  8: syhemvu_generic_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread,  8><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod); break;
				// return error otherwise:
				default: printf("CHEMV-UPPER ERROR: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}
	}
	else if(uplo == 'L' || uplo == 'l')
	{
		/** configuration params **/
		const int chemv_bs = chemv_lower_bs;
		const int thread_x = chemv_bs;
		const int thread_y = chemv_lower_ty;
		const int elements_per_thread = (chemv_bs/(2*thread_y)) ;
		/** end configuration params **/

		int mod = m % chemv_bs;
		int blocks = m / chemv_bs + (mod != 0);
		dim3 dimBlock(thread_x, thread_y);
		dim3 dimGrid(blocks,1);
		dim3 dimGrid_(blocks,chemv_lower_by);

		if(mod == 0)
		{
			syhemvl_special_d<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy);
			syhemvl_special_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid_, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy);
		}
		else
		{
		  	syhemvl_generic_d<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod);
			syhemvl_generic_nd<hipFloatComplex, chemv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid_, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod);
		}
	}
	else{printf("Upper/Lower mode %c is not supported \n", uplo); return -1;}
	return 0;
}

extern "C"
int kblas_chemv( char uplo,
				int m, hipFloatComplex alpha, hipFloatComplex *dA, int lda,
				hipFloatComplex *dX, int incx,
				hipFloatComplex  beta, hipFloatComplex *dY, int incy)
{
	return kblas_chemv_driver( uplo, m, alpha, dA, lda, dX, incx, beta, dY, incy, 0);
}

extern "C"
int kblas_chemv_async( 	char uplo,
						int m, hipFloatComplex alpha, hipFloatComplex *dA, int lda,
						hipFloatComplex *dX, int incx,
						hipFloatComplex  beta, hipFloatComplex *dY, int incy, hipStream_t stream)
{
	return kblas_chemv_driver( uplo, m, alpha, dA, lda, dX, incx, beta, dY, incy, stream);
}
