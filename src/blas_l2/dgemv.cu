/**
 * @copyright (c) 2012- King Abdullah University of Science and
 *                      Technology (KAUST). All rights reserved.
 **/


/**
 * @file src/blas_l2/dgemv.cu

 * KBLAS is a high performance CUDA library for subset of BLAS
 *    and LAPACK routines optimized for NVIDIA GPUs.
 * KBLAS is provided by KAUST.
 *
 * @version 4.0.0
 * @author Ahmad Abdelfattah
 * @date 2020-12-10
 **/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "gemv_core.cuh"

#if(TARGET_SM >= 30)

#define dgemvn_bs		(64)
#define dgemvn_ty		(4)
#define dgemvn_by		(4)

#define dgemvt_bs		(64)
#define dgemvt_ty		(4)
#define dgemvt_by		(2)

#else

#define dgemvn_bs		(64)
#define dgemvn_ty		(8)
#define dgemvn_by		(2)

#define dgemvt_bs		(64)
#define dgemvt_ty		(8)
#define dgemvt_by		(2)

#endif

extern "C"
int kblas_dscal_async(int n, double alpha, double *x, int incx, hipStream_t stream);


int kblas_dgemv_driver(	char trans, int rows, int cols,
						double alpha, double *dA, int lda,
						double *dX, int incx,
						double  beta, double *dY, int incy,
						hipStream_t stream)
{
	if(trans == 'n' || trans == 'N')
	{
		// scaling with beta
		kblas_dscal_async(rows, beta, dY, incy, stream);

		int mod_r = rows % dgemvn_bs;
		int mod_c = cols % dgemvn_bs;

		if(mod_r == 0)
		{
			if(mod_c == 0)
			{
				// special case
				int blocks = rows/dgemvn_bs;
				const int thread_x = dgemvn_bs;
				const int thread_y = dgemvn_ty;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, dgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				gemvn_special<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy);
			}
			else
			{
				// generic case for columns only
				int blocks = rows/dgemvn_bs;
				blocks += 1;	// dummy thread block
				const int thread_x = dgemvn_bs;
				const int thread_y = dgemvn_ty;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, dgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				const int irregular_cols = mod_c % elements_per_thread;
				switch(irregular_cols)
				{
					/**
					 * The kernel for irregular dimensions has an extra template parameter.
				 	 * This parameter must be among the values listed in the switch-case statement below.
				 	 * The possible values are in the range 0 - (elements_per_thread-1)
				 	 * Make sure these values are updated whenever you change the configuration parameters.
					**/
					case  0: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  1: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  2: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  3: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  4: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  5: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  6: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  7: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  8: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					default: printf("DGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;

				}
			}
		}
		else	// mod_r != 0
		{
			if(mod_c == 0)
			{
				// generic case for columns only
				int blocks = (rows/dgemvn_bs) + (mod_r != 0);
				const int thread_x = dgemvn_bs;
				const int thread_y = dgemvn_ty;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, dgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread, 0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c);
			}
			else
			{
				// generic case for rows and cols
				int blocks = (rows/dgemvn_bs) + (mod_r != 0);
				const int thread_x = dgemvn_bs;
				const int thread_y = dgemvn_ty;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, dgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				const int irregular_cols = mod_c % elements_per_thread;
				switch(irregular_cols)
				{
					/**
					 * The kernel for irregular dimensions has an extra template parameter.
				 	 * This parameter must be among the values listed in the switch-case statement below.
				 	 * The possible values are in the range 0 - (elements_per_thread-1)
				 	 * Make sure these values are updated whenever you change the configuration parameters.
					**/
					case  0: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  1: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  2: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  3: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  4: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  5: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  6: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  7: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  8: gemvn_generic<double, dgemvn_bs, dgemvn_bs, dgemvn_ty, elements_per_thread,  8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					default: printf("DGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;

				}
			}
		}
	}	// end of non-transpose case
	else if (trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
	{
		int conj;
		if(trans == 'c' || trans == 'C') conj = 1;
		else conj = 0;
		// scaling with beta
		kblas_dscal_async(cols, beta, dY, incy, stream);

		int mod_r = rows % dgemvt_bs;
		int mod_c = cols % dgemvt_bs;

		if(mod_c == 0)
		{
			if(mod_r == 0)
			{
				// special case
				int blocks = cols/dgemvt_bs;
				const int thread_x = dgemvt_bs;
				const int thread_y = dgemvt_ty;
				const int elements_per_thread = thread_x/(2*thread_y);
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, dgemvt_by);
				gemvt_special<double, dgemvt_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, conj);
			}
			else
			{
				// mod_r != 0
				int blocks = cols/dgemvt_bs;
				blocks += 1;	// dummy thread block
				const int thread_x = dgemvt_bs;
				const int thread_y = dgemvt_ty;
				const int elements_per_thread = thread_x/(2*thread_y);
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, dgemvt_by);
				gemvt_generic<double, dgemvt_bs, thread_x, thread_y, elements_per_thread, 0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj);
			}
		}
		else	// mod_c != 0
		{
			int blocks = cols/dgemvt_bs + (mod_c != 0);
			const int thread_x = dgemvt_bs;
			const int thread_y = dgemvt_ty;
			const int elements_per_thread = thread_x/(2*thread_y);
			const int irregular_cols = mod_c % elements_per_thread;
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, dgemvt_by);

			switch(irregular_cols)
			{
				/**
				  * The kernel for irregular dimensions has an extra template parameter.
				  * This parameter must be among the values listed in the switch-case statement below.
				  * The possible values are in the range 0 - (elements_per_thread-1)
				  * Make sure these values are updated whenever you change the configuration parameters.
				**/
				case  0: gemvt_generic<double, dgemvt_bs, thread_x, thread_y, elements_per_thread,  0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  1: gemvt_generic<double, dgemvt_bs, thread_x, thread_y, elements_per_thread,  1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  2: gemvt_generic<double, dgemvt_bs, thread_x, thread_y, elements_per_thread,  2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  3: gemvt_generic<double, dgemvt_bs, thread_x, thread_y, elements_per_thread,  3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  4: gemvt_generic<double, dgemvt_bs, thread_x, thread_y, elements_per_thread,  4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  5: gemvt_generic<double, dgemvt_bs, thread_x, thread_y, elements_per_thread,  5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  6: gemvt_generic<double, dgemvt_bs, thread_x, thread_y, elements_per_thread,  6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  7: gemvt_generic<double, dgemvt_bs, thread_x, thread_y, elements_per_thread,  7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  8: gemvt_generic<double, dgemvt_bs, thread_x, thread_y, elements_per_thread,  8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				default: printf("DGEMV-T error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}
	}
	else
	{
		printf("DGEMV error: Unrecognized transpose mode %c \n", trans);
		return -1;
	}

	return 0;
}

extern "C"
int kblas_dgemv(char trans, int rows, int cols,
				double alpha, double *dA, int lda,
				double *dX, int incx,
				double  beta, double *dY, int incy)
{
	return kblas_dgemv_driver(	trans, rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, 0);
}

extern "C"
int kblas_dgemv_async(	char trans, int rows, int cols,
						double alpha, double *dA, int lda,
						double *dX, int incx,
						double  beta, double *dY, int incy,
						hipStream_t stream)
{
	return kblas_dgemv_driver(	trans, rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, stream);
}
