#include "hip/hip_runtime.h"
/**
 * @copyright (c) 2012- King Abdullah University of Science and
 *                      Technology (KAUST). All rights reserved.
 **/


/**
 * @file src/blas_l2/dgemv_mgpu.cu

 * KBLAS is a high performance CUDA library for subset of BLAS
 *    and LAPACK routines optimized for NVIDIA GPUs.
 * KBLAS is provided by KAUST.
 *
 * @version 3.0.0
 * @author Ahmad Abdelfattah
 * @date 2018-11-14
 **/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "gemv_mgpu_core.cuh"
#include "gemv_mgpu_offset_core.cuh"
#include "kblas_defs.h"

#if(TARGET_SM >= 30)

#define dgemvn_mgpu_bs		(64)
#define dgemvn_mgpu_ty		(4)
//#define dgemvn_mgpu_by		(2)

#define dgemvt_mgpu_bs		(64)
#define dgemvt_mgpu_ty		(4)
//#define dgemvt_mgpu_by		(2)

#else

#define dgemvn_mgpu_bs		(64)
#define dgemvn_mgpu_ty		(8)
//#define dgemvn_mgpu_by		(1)

#define dgemvt_mgpu_bs		(64)
#define dgemvt_mgpu_ty		(8)
//#define dgemvt_mgpu_by		(1)

#endif

extern "C"
int kblas_dscal_async(int n, double alpha, double *x, int incx, hipStream_t stream);

extern "C"
int kblas_dgemv_mgpu_driver( char trans, int rows, int cols,
						double alpha, double *dA, int lda,
						double *dX, int incx,
						double  beta, double *dY, int incy, int gpu_gid, int ngpus,
						hipStream_t stream = 0)
{
    const double d_zero = 0.0;

	if(trans == 'n' || trans == 'N')
	{
		//******** config parameters
		const int thread_x = dgemvn_mgpu_bs;
		const int thread_y = dgemvn_mgpu_ty;
		const int elements_per_thread = thread_x/(2*thread_y);
		int grid_y_n = 1 * ngpus;
		//**************************

		// scaling with beta
		//if(gpu_gid == 0)hipblasDscal(rows, beta, dY, incy);
		if(gpu_gid == 0)kblas_dscal_async(rows, beta, dY, incy, stream);
		else kblas_dscal_async(rows, d_zero, dY, incy, stream);

		int cols_ = dgemvn_mgpu_bs * ( (cols/dgemvn_mgpu_bs)/ngpus );
		if(gpu_gid < (cols/dgemvn_mgpu_bs)%ngpus) cols_ += dgemvn_mgpu_bs;
		if(gpu_gid == (cols/dgemvn_mgpu_bs)%ngpus) cols_ += cols%dgemvn_mgpu_bs;

		int mod_r = rows % dgemvn_mgpu_bs;
		int mod_c = cols_ % dgemvn_mgpu_bs;

		if(mod_r == 0)
		{
			if(mod_c == 0)
			{
				// special case
				int blocks = rows/dgemvn_mgpu_bs;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_n);
				if(blocks == 0) return 0;
				hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_special<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, gpu_gid, ngpus);
			}
			else
			{
				// generic case for columns only
				const int irregular_cols = mod_c % elements_per_thread;

				int blocks = rows/dgemvn_mgpu_bs;
				blocks += 1;	// dummy thread block
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_n);
				if(blocks == 0) return 0;
				switch(irregular_cols)
				{
					/**
					 * The kernel for irregular dimensions has an extra template parameter.
				 	 * This parameter must be among the values listed in the switch-case statement below.
				 	 * The possible values are in the range 0 - (elements_per_thread-1)
				 	 * Make sure these values are updated whenever you change the configuration parameters.
					**/
					case  0: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  1: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  1>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  2: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  2>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  3: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  3>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  4: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  4>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  5: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  5>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  6: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  6>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  7: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  7>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  8: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  8>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  9: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  9>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 10: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 10>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 11: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 11>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 12: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 12>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 13: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 13>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 14: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 14>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 15: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 15>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					default: printf("DGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;

				}
			}
		}
		else	// mod_r != 0
		{
			if(mod_c == 0)
			{
				// generic case for columns only
				int blocks = (rows/dgemvn_mgpu_bs) + (mod_r != 0);
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_n);
				if(blocks == 0) return 0;
				hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus);
			}
			else
			{
				// generic case for rows and cols
				const int irregular_cols = mod_c % elements_per_thread;

				int blocks = (rows/dgemvn_mgpu_bs) + (mod_r != 0);

				//printf("gpu_gid = %d, cols_ = %d \n", gpu_gid, cols_);
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_n);
				if(blocks == 0) return 0;
				switch(irregular_cols)
				{
					/**
					 * The kernel for irregular dimensions has an extra template parameter.
				 	 * This parameter must be among the values listed in the switch-case statement below.
				 	 * The possible values are in the range 0 - (elements_per_thread-1)
				 	 * Make sure these values are updated whenever you change the configuration parameters.
					**/
					case  0: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  1: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  1>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  2: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  2>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  3: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  3>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  4: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  4>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  5: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  5>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  6: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  6>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  7: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  7>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  8: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  8>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  9: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  9>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 10: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 10>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 11: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 11>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 12: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 12>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 13: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 13>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 14: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 14>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 15: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 15>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					default: printf("DGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;

				}
			}
		}
	}	// end of non-transpose case
	else if (trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
	{
		int conj;
		if(trans == 'c' || trans == 'C') conj = 1;
		else conj = 0;
		//************ config parameters
		const int thread_x = dgemvt_mgpu_bs;
		const int thread_y = dgemvt_mgpu_ty;
		const int elements_per_thread = thread_x/(2*thread_y);
		int grid_y_t = 1 * ngpus;
		//******************************

		// scaling with beta
		//if(gpu_gid == 0)hipblasDscal(cols, beta, dY, incy);
		if(gpu_gid == 0)kblas_dscal_async(cols, beta, dY, incy, stream);
		else kblas_dscal_async(cols, d_zero, dY, incy, stream);

		int cols_ = dgemvt_mgpu_bs * ( (cols/dgemvt_mgpu_bs)/ngpus );
		if(gpu_gid < (cols/dgemvt_mgpu_bs)%ngpus) cols_ += dgemvt_mgpu_bs;
		if(gpu_gid == (cols/dgemvt_mgpu_bs)%ngpus) cols_ += cols%dgemvt_mgpu_bs;

		int mod_r = rows % dgemvt_mgpu_bs;
		int mod_c = cols_ % dgemvt_mgpu_bs;

		if(mod_c == 0)
		{
			if(mod_r == 0)
			{
				// special case
				int blocks = cols_/dgemvt_mgpu_bs;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_t);
				if(blocks == 0) return 0;
				hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_special<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, gpu_gid, ngpus, conj);
			}
			else
			{
				// mod_r != 0
				int blocks = cols_/dgemvt_mgpu_bs;
				blocks += 1;	// dummy thread block
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_t);
				if(blocks == 0) return 0;
				hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj);
			}
		}
		else	// mod_c != 0
		{
			const int irregular_cols = mod_c % elements_per_thread;

			int blocks = cols_/dgemvt_mgpu_bs + (mod_c != 0);
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_t);
			if(blocks == 0) return 0;

			switch(irregular_cols)
			{
				/**
				  * The kernel for irregular dimensions has an extra template parameter.
				  * This parameter must be among the values listed in the switch-case statement below.
				  * The possible values are in the range 0 - (elements_per_thread-1)
				  * Make sure these values are updated whenever you change the configuration parameters.
				**/
				case  0: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  1: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  1>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  2: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  2>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  3: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  3>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  4: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  4>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  5: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  5>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  6: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  6>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  7: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  7>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  8: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  8>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  9: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  9>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 10: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 10>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 11: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 11>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 12: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 12>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 13: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 13>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 14: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 14>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 15: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 15>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				default: printf("DGEMV-T error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}
	}
	else
	{
		printf("DGEMV error: Unrecognized transpose mode %c \n", trans);
		return -1;
	}

	return 0;
}
/*************************************************************************************/
extern "C"
int kblas_dgemv_mgpu_driver_offset( char trans, int rows, int cols,
						double alpha, double *dA, int lda,
						double *dX, int incx,
						double  beta, double *dY, int incy, int gpu_gid, int ngpus,
						int offset_r, int offset_c,
						hipStream_t stream = 0)
{
    const double d_zero = 0.0;
	if(trans == 'n' || trans == 'N')
	{

		//**** Config parameters
		const int thread_x = dgemvn_mgpu_bs;
		const int thread_y = dgemvn_mgpu_ty;
		const int elements_per_thread = thread_x/(2*thread_y);
		const int grid_y_n = 2 * ngpus;
		//*************************

		/** offset necessary calculation **/
		int offset_r_ = offset_r % dgemvn_mgpu_bs;
		int offset_c_ = offset_c % dgemvn_mgpu_bs;
		int total_blocks_skipped_r = offset_r / dgemvn_mgpu_bs;
		int total_blocks_skipped_c = offset_c / dgemvn_mgpu_bs;
		int my_skipped_blocks_r = total_blocks_skipped_r;
		int my_skipped_blocks_c = total_blocks_skipped_c/ngpus;
		if(gpu_gid < (total_blocks_skipped_c%ngpus)) my_skipped_blocks_c += 1;
		int ref_gpu = total_blocks_skipped_c%ngpus;
		int new_gpu_gid = (gpu_gid - ref_gpu + ngpus) % ngpus;
		// Advance pointers accordingly
		dA += my_skipped_blocks_c * dgemvn_mgpu_bs * lda;
		dA += total_blocks_skipped_r * dgemvn_mgpu_bs;
		dX += total_blocks_skipped_c * dgemvn_mgpu_bs * incx;
		dY += my_skipped_blocks_r * dgemvn_mgpu_bs * incy;
		rows -= total_blocks_skipped_r * dgemvn_mgpu_bs;
		cols -= total_blocks_skipped_c * dgemvn_mgpu_bs;
		/** end offset necessary calculation **/

		int nstripes = (cols/dgemvn_mgpu_bs) + ((cols%dgemvn_mgpu_bs) != 0);

		// scaling with beta
		if(gpu_gid == 0)kblas_dscal_async(rows-offset_r_, beta, dY+(offset_r_*incy), incy, stream);
		else kblas_dscal_async(rows-offset_r_, d_zero, dY+(offset_r_*incy), incy, stream);

		int cols_ = dgemvn_mgpu_bs * ( (cols/dgemvn_mgpu_bs)/ngpus );
		if(new_gpu_gid < (cols/dgemvn_mgpu_bs)%ngpus) cols_ += dgemvn_mgpu_bs;
		if(new_gpu_gid == (cols/dgemvn_mgpu_bs)%ngpus) cols_ += cols%dgemvn_mgpu_bs;

		int mod_r = rows % dgemvn_mgpu_bs;
		int mod_c = cols_ % dgemvn_mgpu_bs;

		if(mod_r == 0 && mod_c == 0)
		{
			// special case
			int blocks = rows/dgemvn_mgpu_bs;
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_n);
			if(blocks == 0) return 0;
			hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_special_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_);
		}
		else
		{
			// generic case for columns only
			const int irregular_cols = mod_c % elements_per_thread;

			int blocks = (rows/dgemvn_mgpu_bs) + (mod_r != 0);
			if(mod_r == 0)blocks += 1;	// dummy thread block, will return immediately if mod_r == 0

			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_n);
			if(blocks == 0) return 0;

			switch(irregular_cols)
			{
				/**
				 * The kernel for irregular dimensions has an extra template parameter.
				 * This parameter must be among the values listed in the switch-case statement below.
				 * The possible values are in the range 0 - (elements_per_thread-1)
				 * Make sure these values are updated whenever you change the configuration parameters.
				**/
				case  0: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  1: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  1>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  2: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  2>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  3: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  3>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  4: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  4>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  5: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  5>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  6: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  6>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  7: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  7>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  8: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  8>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  9: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread,  9>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 10: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 10>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 11: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 11>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 12: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 12>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 13: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 13>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 14: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 14>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 15: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<double, dgemvn_mgpu_bs, dgemvn_mgpu_bs, dgemvn_mgpu_ty, elements_per_thread, 15>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				default: printf("DGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}
	}	// end of non-transpose case
	else if (trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
	{
		int conj;
		if(trans == 'c' || trans == 'C') conj = 1;
		else conj = 0;
		//**** Config parameters
		const int thread_x = dgemvt_mgpu_bs;
		const int thread_y = dgemvt_mgpu_ty;
		const int elements_per_thread = thread_x/(2*thread_y);
		const int grid_y_t = 2 * ngpus;
		//*************************

		/** offset necessary calculation **/
		int offset_r_ = offset_r % dgemvt_mgpu_bs;
		int offset_c_ = offset_c % dgemvt_mgpu_bs;
		int total_blocks_skipped_r = offset_r / dgemvt_mgpu_bs;
		int total_blocks_skipped_c = offset_c / dgemvt_mgpu_bs;
		int my_skipped_blocks_r = total_blocks_skipped_r;
		int my_skipped_blocks_c = total_blocks_skipped_c/ngpus;
		if(gpu_gid < (total_blocks_skipped_c%ngpus)) my_skipped_blocks_c += 1;
		int ref_gpu = total_blocks_skipped_c%ngpus;
		int new_gpu_gid = (gpu_gid - ref_gpu + ngpus) % ngpus;

		//if(new_gpu_gid != 3){return 0;}
		// Advance pointers accordingly
		dA += my_skipped_blocks_c * dgemvt_mgpu_bs * lda;
		dA += my_skipped_blocks_r * dgemvt_mgpu_bs;
		dX += total_blocks_skipped_r * dgemvt_mgpu_bs * incx;
		dY += total_blocks_skipped_c * dgemvt_mgpu_bs * incy;
		rows -= total_blocks_skipped_r * dgemvt_mgpu_bs;
		cols -= total_blocks_skipped_c * dgemvt_mgpu_bs;
		/** end offset necessary calculation **/

		int nstripes = (cols/dgemvt_mgpu_bs) + ((cols%dgemvt_mgpu_bs) != 0);

		// scaling with beta
		//if(gpu_gid == 0)hipblasDscal(cols-offset_, beta, dY+(offset_*incy), incy);
		if(gpu_gid == 0)kblas_dscal_async(cols-offset_c_, beta, dY+(offset_c_*incy), incy, stream);
		else kblas_dscal_async(cols-offset_r_, d_zero, dY+(offset_r_*incy), incy, stream);

		int cols_ = dgemvt_mgpu_bs * ( (cols/dgemvt_mgpu_bs)/ngpus );
		if(new_gpu_gid < (cols/dgemvt_mgpu_bs)%ngpus) cols_ += dgemvt_mgpu_bs;
		if(new_gpu_gid == (cols/dgemvt_mgpu_bs)%ngpus) cols_ += cols%dgemvt_mgpu_bs;

		int mod_r = rows % dgemvt_mgpu_bs;
		int mod_c = cols_ % dgemvt_mgpu_bs;

		if(mod_r == 0 && mod_c == 0)
		{
			int blocks = cols_/dgemvt_mgpu_bs;
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_t);
			if(blocks == 0) return 0;
			hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_special_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj);

		}
		else
		{
			const int irregular_cols = mod_c % elements_per_thread;
			int blocks = cols_/dgemvt_mgpu_bs + (mod_c != 0);
			int gpu_last = (nstripes+ngpus-1)%ngpus;
			if(mod_c == 0 && new_gpu_gid == gpu_last) blocks += 1; // dummy thread block, will return if mod_c == 0
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_t);
			if(blocks == 0) return 0;

			switch(irregular_cols)
			{
				/**
				  * The kernel for irregular dimensions has an extra template parameter.
				  * This parameter must be among the values listed in the switch-case statement below.
				  * The possible values are in the range 0 - (elements_per_thread-1)
				  * Make sure these values are updated whenever you change the configuration parameters.
				**/
				case  0: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  1: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  1>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  2: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  2>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  3: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  3>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  4: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  4>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  5: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  5>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  6: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  6>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  7: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  7>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  8: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  8>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  9: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  9>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 10: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 10>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 11: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 11>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 12: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 12>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 13: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 13>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 14: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 14>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 15: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<double, dgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 15>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				default: printf("DGEMV-T error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}
	}
	else
	{
		printf("DGEMV error: Unrecognized transpose mode %c \n", trans);
		return -1;
	}

	return 0;
}
/***********************************************************************************/
extern "C"
int kblas_dgemv_mgpu( char trans, int rows, int cols,
						double alpha, double **dA, int lda,
						double **dX, int incx,
						double  beta, double **dY, int incy,
						int ngpus,
						int offset_r, int offset_c)
{
    const int ngpus_local = ngpus;
	if(offset_r == 0 && offset_c == 0)
	{
		for(int i = 0; i < ngpus_local; i++)
		{
			hipSetDevice(gpu_lid[i]);
			kblas_dgemv_mgpu_driver(trans, rows, cols, alpha, dA[i], lda, dX[i], incx, beta, dY[i], incy, gpu_gid[i], ngpus);
		}
	}
	else
	{
		for(int i = 0; i < ngpus_local; i++)
		{
			hipSetDevice(gpu_lid[i]);
			kblas_dgemv_mgpu_driver_offset(trans, rows, cols, alpha, dA[i], lda, dX[i], incx, beta, dY[i], incy, gpu_gid[i], ngpus, offset_r, offset_c);
		}
	}

	// wait for gpus to finish
	for(int i = 0; i < ngpus_local; i++)
	{
		hipSetDevice(gpu_lid[i]);
		hipDeviceSynchronize();
	}
	return 0;
}

/*************************************************************************************/
extern "C"
int kblas_dgemv_mgpu_async( char trans, int rows, int cols,
							double alpha, double **dA, int lda,
							double **dX, int incx,
							double  beta, double **dY, int incy,
							int ngpus,
							int offset_r, int offset_c,
							hipStream_t stream[MAX_NGPUS][MAX_STREAMS])
{
    const int ngpus_local = ngpus;
	if(offset_r == 0 && offset_c == 0)
	{
		for(int i = 0; i < ngpus_local; i++)
		{
			hipSetDevice(gpu_lid[i]);
			kblas_dgemv_mgpu_driver(trans, rows, cols, alpha, dA[i], lda, dX[i], incx, beta, dY[i], incy, gpu_gid[i], ngpus, stream[i][0]);
		}
	}
	else
	{
		for(int i = 0; i < ngpus_local; i++)
		{
			hipSetDevice(gpu_lid[i]);
			kblas_dgemv_mgpu_driver_offset(trans, rows, cols, alpha, dA[i], lda, dX[i], incx, beta, dY[i], incy, gpu_gid[i], ngpus, offset_r, offset_c);
		}
	}

	return 0;
}
/*************************************************************************************/

extern "C"
int get_dgemv_mgpu_bs(char trans)
{
    if(trans == 'n' || trans == 'N')
        return dgemvn_mgpu_bs;
    else if (trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
        return dgemvt_mgpu_bs;
    else
        {printf("Error ..  input %c is not supported for gemv \n", trans); return -1;}
}
