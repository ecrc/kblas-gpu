/**
 * @copyright (c) 2012- King Abdullah University of Science and
 *                      Technology (KAUST). All rights reserved.
 **/


/**
 * @file src/blas_l2/dscal.cu

 * KBLAS is a high performance CUDA library for subset of BLAS
 *    and LAPACK routines optimized for NVIDIA GPUs.
 * KBLAS is provided by KAUST.
 *
 * @version 4.0.0
 * @author Ahmad Abdelfattah
 * @date 2020-12-10
 **/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "scal_core.cuh"

#define dscal_nbx		(128)

int kblas_dscal_driver(int n, double alpha, double *x, int incx, hipStream_t stream)
{
	int gridx = n / dscal_nbx + (n % dscal_nbx != 0);

	dim3 dimBlock(dscal_nbx, 1);
	dim3 dimGrid(gridx, 1);

	scal<double><<<dimGrid, dimBlock, 0, stream>>>(n, alpha, x, incx);

	return 0;
}

extern "C"
int kblas_dscal(int n, double alpha, double *x, int incx)
{
	return kblas_dscal_driver(n, alpha, x, incx, 0);
}

extern "C"
int kblas_dscal_async(int n, double alpha, double *x, int incx, hipStream_t stream)
{
	return kblas_dscal_driver(n, alpha, x, incx, stream);
}
