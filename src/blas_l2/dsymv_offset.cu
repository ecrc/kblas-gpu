/**
 * @copyright (c) 2012- King Abdullah University of Science and
 *                      Technology (KAUST). All rights reserved.
 **/


/**
 * @file src/blas_l2/dsymv_offset.cu

 * KBLAS is a high performance CUDA library for subset of BLAS
 *    and LAPACK routines optimized for NVIDIA GPUs.
 * KBLAS is provided by KAUST.
 *
 * @version 4.0.0
 * @author Ahmad Abdelfattah
 * @date 2020-12-10
 **/

#include "syhemv_offset_core.cuh"

#if(TARGET_SM >= 30)

#define dsymv_upper_bs	(32)
#define dsymv_upper_ty	(2)
#define dsymv_upper_by  (4)

#define dsymv_lower_bs	(32)
#define dsymv_lower_ty	(2)
#define dsymv_lower_by  (4)

#else

#define dsymv_upper_bs	(64)
#define dsymv_upper_ty	(8)
#define dsymv_upper_by  (2)

#define dsymv_lower_bs	(64)
#define dsymv_lower_ty	(8)
#define dsymv_lower_by  (2)

#endif

/*************************************************************************************/
int kblas_dsymv_offset_driver(  char uplo, int m,
							    double alpha, double *dA, int lda,
							    double *dX, int incx,
							    double  beta, double *dY, int incy,
						    	int offset,
							    hipStream_t stream = 0)
{
	// handle the case when incx and/or incy is -ve
	if(incx < 0) dX -= (m-1) * incx;
	if(incy < 0) dY -= (m-1) * incy;

	if(uplo == 'U' || uplo == 'u')
	{
		/** configuration params **/
		const int dsymv_bs = dsymv_upper_bs;
		const int thread_x = dsymv_bs;
		const int thread_y = dsymv_upper_ty;
		const int elements_per_thread = (dsymv_bs/(2*thread_y)) ;
		/** end configuration params **/

		/** offset necessary calculation **/
		int offset_ = offset % dsymv_bs;
		int total_blocks_skipped = offset / dsymv_bs;
		int my_skipped_blocks = total_blocks_skipped/ngpus;
		if(gpu_gid < (total_blocks_skipped%ngpus)) my_skipped_blocks += 1;
		int ref_gpu = total_blocks_skipped%ngpus;
		int new_gpu_gid = (gpu_gid - ref_gpu + ngpus) % ngpus;
		// Advance pointers accordingly
		dA += my_skipped_blocks * dsymv_bs * lda;
		dA += total_blocks_skipped * dsymv_bs;
		dX += total_blocks_skipped * dsymv_bs * incx;
		dY += total_blocks_skipped * dsymv_bs * incy;
		m  -= total_blocks_skipped * dsymv_bs;
		/** end offset necessary calculation **/

		int mod = m % dsymv_bs;
		int nstripes = m / dsymv_bs + (mod != 0);
		int blocks = nstripes/ngpus;
		if(new_gpu_gid < (nstripes%ngpus) ) blocks += 1;
		dim3 dimBlock(thread_x, thread_y);
		dim3 dimGrid(blocks,1);
		dim3 dimGrid_(blocks, dsymv_upper_by);

		if(blocks == 0) return 0;

		if(mod == 0)
		{
			syhemvu_special_d_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy, nstripes, offset_);
			syhemvu_special_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid_, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy, nstripes, offset_);
		}
		else
		{
			syhemvu_generic_d_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy, ngpus, nstripes, offset_);
			const int irregular_part = mod % elements_per_thread;
			/**
			 * The upper case kernel for irregular dimensions has an extra template parameter.
			 * This parameter must be among the values listed in the switch-case statement below.
			 * The possible values are in the range 0 - (elements_per_thread-1)
			 * Make sure these values are updated whenever you change the configuration parameters.
			 **/
			switch(irregular_part)
			{
				case  0: syhemvu_generic_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread,  0><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod, nstripes, offset_); break;
				case  1: syhemvu_generic_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread,  1><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod, nstripes, offset_); break;
				case  2: syhemvu_generic_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread,  2><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod, nstripes, offset_); break;
				case  3: syhemvu_generic_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread,  3><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod, nstripes, offset_); break;
				case  4: syhemvu_generic_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread,  4><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod, nstripes, offset_); break;
				case  5: syhemvu_generic_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread,  5><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod, nstripes, offset_); break;
				case  6: syhemvu_generic_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread,  6><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod, nstripes, offset_); break;
				case  7: syhemvu_generic_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread,  7><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod, nstripes, offset_); break;
				case  8: syhemvu_generic_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread,  8><<<dimGrid_, dimBlock, 0, stream>>>( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod, nstripes, offset_); break;
				// return error otherwise:
				default: printf("DSYMV-UPPER ERROR: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}

	}
	else if(uplo == 'L' || uplo == 'l')
	{
		/** configuration params **/
		const int dsymv_bs = dsymv_lower_bs;
		const int thread_x = dsymv_bs;
		const int thread_y = dsymv_lower_ty;
		const int elements_per_thread = (dsymv_bs/(2*thread_y)) ;
		/** end configuration params **/

		/** offset necessary calculation **/
		int offset_ = offset % dsymv_bs;
		int total_blocks_skipped = offset / dsymv_bs;
		int my_skipped_blocks = total_blocks_skipped/ngpus;
		if(gpu_gid < (total_blocks_skipped%ngpus)) my_skipped_blocks += 1;
		int ref_gpu = total_blocks_skipped%ngpus;
		int new_gpu_gid = (gpu_gid - ref_gpu + ngpus) % ngpus;
		// Advance pointers accordingly
		dA += my_skipped_blocks * dsymv_bs * lda;
		dA += total_blocks_skipped * dsymv_bs;
		dX += total_blocks_skipped * dsymv_bs * incx;
		dY += total_blocks_skipped * dsymv_bs * incy;
		m  -= total_blocks_skipped * dsymv_bs;
		/** end offset necessary calculation **/

		int mod = m % dsymv_bs;
		int nstripes = m / dsymv_bs + (mod != 0);
		int blocks = nstripes/ngpus;
		if(new_gpu_gid < (nstripes%ngpus) ) blocks += 1;
		dim3 dimBlock(thread_x, thread_y);
		dim3 dimGrid(blocks,1);
		dim3 dimGrid_(blocks, dsymv_lower_by);

		if(blocks == 0) return 0;

		if(mod == 0)
		{
			syhemvl_special_d_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy, nstripes, offset_);
			syhemvl_special_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid_, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy, nstripes, offset_);
		}
		else
		{
		  	syhemvl_generic_d_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod, nstripes, offset_);
			syhemvl_generic_nd_offset<double, dsymv_bs, thread_x, thread_y, elements_per_thread><<<dimGrid_, dimBlock, 0, stream>>> ( m, alpha, dA, lda, dX, incx, beta, dY, incy, mod, nstripes, offset_);
		}
	}
	else{printf("Upper/Lower mode %c is not supported \n", uplo); return -1;}
	return 0;
}

/*************************************************************************************/
extern "C"
int kblas_dsymv_offset( char uplo, int m,
						double alpha, double *dA, int lda,
						double *dX, int incx,
						double beta, double *dY, int incy,
						int offset)
{
	return kblas_dsymv_offset_driver(uplo, m, alpha, dA, lda, dX, incx, beta, dY, incy, offset);
}
/*************************************************************************************/
extern "C"
int kblas_dsymv_offset_async( char uplo, int m,
							double alpha, double *dA, int lda,
							double *dX, int incx,
							double  beta, double *dY, int incy,
							int offset,
							hipStream_t stream)
{
    return kblas_dsymv_offset_driver(uplo, m, alpha, dA, lda, dX, incx, beta, dY, incy, offset, stream);
}
/*************************************************************************************/
