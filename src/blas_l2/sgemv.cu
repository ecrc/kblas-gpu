/**
 * @copyright (c) 2012- King Abdullah University of Science and
 *                      Technology (KAUST). All rights reserved.
 **/


/**
 * @file src/blas_l2/sgemv.cu

 * KBLAS is a high performance CUDA library for subset of BLAS
 *    and LAPACK routines optimized for NVIDIA GPUs.
 * KBLAS is provided by KAUST.
 *
 * @version 4.0.0
 * @author Ahmad Abdelfattah
 * @date 2020-12-10
 **/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "gemv_core.cuh"

#if(TARGET_SM >= 30)

#define sgemvn_bs		(64)
#define sgemvn_ty		(8)
#define sgemvn_by		(8)

#define sgemvt_bs		(64)
#define sgemvt_ty		(4)
#define sgemvt_by		(4)

#else

#define sgemvn_bs		(32)
#define sgemvn_ty		(8)
#define sgemvn_by		(1)

#define sgemvt_bs		(32)
#define sgemvt_ty		(8)
#define sgemvt_by		(1)

#endif

extern "C"
int kblas_sscal_async(int n, float alpha, float *x, int incx, hipStream_t stream);

int kblas_sgemv_driver( char trans, int rows, int cols,
						float alpha, float *dA, int lda,
						float *dX, int incx,
						float  beta, float *dY, int incy, hipStream_t stream)
{
	if(trans == 'n' || trans == 'N')
	{
		// scaling with beta
		kblas_sscal_async(rows, beta, dY, incy, stream);

		int mod_r = rows % sgemvn_bs;
		int mod_c = cols % sgemvn_bs;

		if(mod_r == 0)
		{
			if(mod_c == 0)
			{
				// special case
				int blocks = rows/sgemvn_bs;
				const int thread_x = sgemvn_bs;
				const int thread_y = sgemvn_ty;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				gemvn_special<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy);
				//gemvn_special_<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy);
			}
			else
			{
				// generic case for columns only
				int blocks = rows/sgemvn_bs;
				blocks += 1;	// dummy thread block
				const int thread_x = sgemvn_bs;
				const int thread_y = sgemvn_ty;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				const int irregular_cols = mod_c % elements_per_thread;
				switch(irregular_cols)
				{
					/**
					 * The kernel for irregular dimensions has an extra template parameter.
				 	 * This parameter must be among the values listed in the switch-case statement below.
				 	 * The possible values are in the range 0 - (elements_per_thread-1)
				 	 * Make sure these values are updated whenever you change the configuration parameters.
					**/
					case  0: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  1: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  2: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  3: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  4: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  5: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  6: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  7: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  8: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					default: printf("SGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;

				}
			}
		}
		else	// mod_r != 0
		{
			if(mod_c == 0)
			{
				// generic case for columns only
				int blocks = (rows/sgemvn_bs) + (mod_r != 0);
				const int thread_x = sgemvn_bs;
				const int thread_y = sgemvn_ty;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread, 0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c);
			}
			else
			{
				// generic case for rows and cols
				int blocks = (rows/sgemvn_bs) + (mod_r != 0);
				const int thread_x = sgemvn_bs;
				const int thread_y = sgemvn_ty;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvn_by);
				const int elements_per_thread = thread_x/(2*thread_y);
				const int irregular_cols = mod_c % elements_per_thread;
				switch(irregular_cols)
				{
					/**
					 * The kernel for irregular dimensions has an extra template parameter.
				 	 * This parameter must be among the values listed in the switch-case statement below.
				 	 * The possible values are in the range 0 - (elements_per_thread-1)
				 	 * Make sure these values are updated whenever you change the configuration parameters.
					**/
					case  0: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  1: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  2: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  3: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  4: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  5: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  6: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  7: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					case  8: gemvn_generic<float, sgemvn_bs, sgemvn_bs, sgemvn_ty, elements_per_thread,  8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c); break;
					default: printf("SGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;

				}
			}
		}
	}	// end of non-transpose case
	else if (trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
	{
		int conj;
		if(trans == 'c' || trans == 'C') conj = 1;
		else conj = 0;
		// scaling with beta
		kblas_sscal_async(cols, beta, dY, incy, stream);

		int mod_r = rows % sgemvt_bs;
		int mod_c = cols % sgemvt_bs;

		if(mod_c == 0)
		{
			if(mod_r == 0)
			{
				// special case
				int blocks = cols/sgemvt_bs;
				const int thread_x = sgemvt_bs;
				const int thread_y = sgemvt_ty;
				const int elements_per_thread = thread_x/(2*thread_y);
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvt_by);
				gemvt_special<float, sgemvt_bs, thread_x, thread_y, elements_per_thread><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, conj);
			}
			else
			{
				// mod_r != 0
				int blocks = cols/sgemvt_bs;
				blocks += 1;	// dummy thread block
				const int thread_x = sgemvt_bs;
				const int thread_y = sgemvt_ty;
				const int elements_per_thread = thread_x/(2*thread_y);
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, sgemvt_by);
				gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread, 0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj);
			}
		}
		else	// mod_c != 0
		{
			int blocks = cols/sgemvt_bs + (mod_c != 0);
			const int thread_x = sgemvt_bs;
			const int thread_y = sgemvt_ty;
			const int elements_per_thread = thread_x/(2*thread_y);
			const int irregular_cols = mod_c % elements_per_thread;
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, sgemvt_by);

			switch(irregular_cols)
			{
				/**
				  * The kernel for irregular dimensions has an extra template parameter.
				  * This parameter must be among the values listed in the switch-case statement below.
				  * The possible values are in the range 0 - (elements_per_thread-1)
				  * Make sure these values are updated whenever you change the configuration parameters.
				**/
				case  0: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  0><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  1: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  1><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  2: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  2><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  3: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  3><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  4: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  4><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  5: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  5><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  6: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  6><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  7: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  7><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				case  8: gemvt_generic<float, sgemvt_bs, thread_x, thread_y, elements_per_thread,  8><<<dimGrid, dimBlock, 0, stream>>>(rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, conj); break;
				default: printf("SGEMV-T error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}
	}
	else
	{
		printf("SGEMV error: Unrecognized transpose mode %c \n", trans);
		return -1;
	}

	return 0;
}

extern "C"
int kblas_sgemv(char trans, int rows, int cols,
				float alpha, float *dA, int lda,
				float *dX, int incx,
				float  beta, float *dY, int incy)
{
	return kblas_sgemv_driver( trans, rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, 0);
}

extern "C"
int kblas_sgemv_async( 	char trans, int rows, int cols,
						float alpha, float *dA, int lda,
						float *dX, int incx,
						float  beta, float *dY, int incy,
						hipStream_t stream)
{
	return kblas_sgemv_driver( trans, rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, stream);
}
