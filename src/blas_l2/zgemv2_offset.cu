/**
 * @copyright (c) 2012- King Abdullah University of Science and
 *                      Technology (KAUST). All rights reserved.
 **/


/**
 * @file src/blas_l2/zgemv2_offset.cu

 * KBLAS is a high performance CUDA library for subset of BLAS
 *    and LAPACK routines optimized for NVIDIA GPUs.
 * KBLAS is provided by KAUST.
 *
 * @version 4.0.0
 * @author Ahmad Abdelfattah
 * @date 2020-12-10
 **/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "gemv2_offset_core.cuh"

#if(TARGET_SM >= 30)

#define zgemvn_offset_nb               	(32)
#define zgemvn_offset_ntcol    			(4)
#define zgemvn_offset_ept              	(2)
#define zgemvn_offset_width    			(zgemvn_offset_ntcol*zgemvn_offset_ept)
#define zgemvn_offset_by               	(4)

#define zgemvt_offset_nb               	(32)
#define zgemvt_offset_ntcol            	(4)
#define zgemvt_offset_ept              	(4)
#define zgemvt_offset_width    			(zgemvt_offset_ntcol*zgemvt_offset_ept)
#define zgemvt_offset_by               	(4)

#else

#define zgemvn_offset_nb               	(64)
#define zgemvn_offset_ntcol    			(8)
#define zgemvn_offset_ept              	(2)
#define zgemvn_offset_width    			(zgemvn_offset_ntcol*zgemvn_offset_ept)
#define zgemvn_offset_by				(1)

#define zgemvt_offset_nb               	(64)
#define zgemvt_offset_ntcol    			(8)
#define zgemvt_offset_ept              	(2)
#define zgemvt_offset_width    			(zgemvt_offset_ntcol*zgemvt_offset_ept)
#define zgemvt_offset_by               	(1)
#endif


extern "C"
int kblas_zscal_async(int n, hipDoubleComplex alpha, hipDoubleComplex *x, int incx, hipStream_t stream);


int kblas_zgemv2_offset_driver(char trans, int rows, int cols,
						hipDoubleComplex alpha, hipDoubleComplex *dA, int lda,
						hipDoubleComplex *dX, int incx,
						hipDoubleComplex  beta, hipDoubleComplex *dY, int incy,
						int offset_r, int offset_c,
						hipStream_t stream)
{
	if(trans == 'n' || trans == 'N')
	{
		// offset necessary calculations
		int offset_r_ = offset_r % zgemvn_offset_nb;
		int offset_c_ = offset_c % zgemvn_offset_width;
		int rows_ = rows - (offset_r - offset_r_);
		int cols_ = cols - (offset_c - offset_c_);

		// Advance pointers
		dA += (offset_c - offset_c_) * lda + (offset_r - offset_r_);
		dX += (offset_c - offset_c_) * incx;
		dY += (offset_r - offset_r_) * incy;

		// scaling with beta
		kblas_zscal_async(rows_, beta, dY, incy, stream);

		int mod_r = rows_ % zgemvn_offset_nb;
		int mod_c = cols_ % zgemvn_offset_width;

		int blocks = rows_/zgemvn_offset_nb;
		if(mod_r != 0) blocks += 1;

		const int thread_x = zgemvn_offset_nb;
		const int thread_y = zgemvn_offset_ntcol;
		const int ept = zgemvn_offset_ept;

		int threshold = mod_c / ept;
		int ept_ = mod_c % ept;
		dim3 dimBlock(thread_x, thread_y);
		dim3 dimGrid(blocks, zgemvn_offset_by);
		//printf("rows_ = %d - cols_ = %d - mod_r = %d - mod_c = %d - offset_r_ = %d - offset_c_ = %d \n", rows_, cols_, mod_r, mod_c, offset_r_, offset_c_);
		switch(ept_)
		{
			case 0: gemvn_offset<hipDoubleComplex, zgemvn_offset_nb, zgemvn_offset_ntcol, ept, zgemvn_offset_width, 0><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, offset_r_, offset_c_); break;
			case 1: gemvn_offset<hipDoubleComplex, zgemvn_offset_nb, zgemvn_offset_ntcol, ept, zgemvn_offset_width, 1><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, offset_r_, offset_c_); break;
			case 2: gemvn_offset<hipDoubleComplex, zgemvn_offset_nb, zgemvn_offset_ntcol, ept, zgemvn_offset_width, 2><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, offset_r_, offset_c_); break;
			case 3: gemvn_offset<hipDoubleComplex, zgemvn_offset_nb, zgemvn_offset_ntcol, ept, zgemvn_offset_width, 3><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, offset_r_, offset_c_); break;
			case 4: gemvn_offset<hipDoubleComplex, zgemvn_offset_nb, zgemvn_offset_ntcol, ept, zgemvn_offset_width, 4><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, offset_r_, offset_c_); break;
			case 5: gemvn_offset<hipDoubleComplex, zgemvn_offset_nb, zgemvn_offset_ntcol, ept, zgemvn_offset_width, 5><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, offset_r_, offset_c_); break;
			case 6: gemvn_offset<hipDoubleComplex, zgemvn_offset_nb, zgemvn_offset_ntcol, ept, zgemvn_offset_width, 6><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, offset_r_, offset_c_); break;
			case 7: gemvn_offset<hipDoubleComplex, zgemvn_offset_nb, zgemvn_offset_ntcol, ept, zgemvn_offset_width, 7><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, offset_r_, offset_c_); break;
			case 8: gemvn_offset<hipDoubleComplex, zgemvn_offset_nb, zgemvn_offset_ntcol, ept, zgemvn_offset_width, 8><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, offset_r_, offset_c_); break;
			default: printf("irregular part %d is not supported, please extend the case statement of zgemv\n", ept_); exit(1);
		}
	}	// end of non-transpose case
	else if(trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
	{
		// offset necessary calculations
		int offset_r_ = offset_r % zgemvt_offset_nb;
		int offset_c_ = offset_c % zgemvt_offset_width;
		int rows_ = rows - (offset_r - offset_r_);
		int cols_ = cols - (offset_c - offset_c_);

		// Advance pointers
		dA += (offset_c - offset_c_) * lda + (offset_r - offset_r_);
		dX += (offset_r - offset_r_) * incx;
		dY += (offset_c - offset_c_) * incy;

		// scaling with beta
		kblas_zscal_async(cols_, beta, dY, incy, stream);

		int mod_r = rows_ % zgemvt_offset_nb;
		int mod_c = cols_ % zgemvt_offset_width;

		int blocks = cols_/zgemvt_offset_width;
		if(mod_c != 0) blocks += 1;

		const int thread_x = zgemvt_offset_nb;
		const int thread_y = zgemvt_offset_ntcol;
		const int ept = zgemvt_offset_ept;

		int threshold = mod_c / ept;
		int ept_ = mod_c % ept;

		dim3 dimBlock(thread_x, thread_y);
		dim3 dimGrid(blocks, zgemvt_offset_by);

		int conj;
		if(trans == 'c' || trans == 'C')conj = 1;
		else conj = 0;
		//printf("modr = %d, modc = %d, threshold = %d, ept_ = %d \n", mod_r, mod_c, threshold, ept_);
		//printf("rows_ = %d - cols_ = %d - mod_r = %d - mod_c = %d - offset_r_ = %d - offset_c_ = %d \n", rows_, cols_, mod_r, mod_c, offset_r_, offset_c_);
		switch(ept_)
		{
			case 0: gemvt_offset<hipDoubleComplex, zgemvt_offset_nb, zgemvt_offset_ntcol, ept, zgemvt_offset_width, 0><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj, offset_r_, offset_c_); break;
			case 1: gemvt_offset<hipDoubleComplex, zgemvt_offset_nb, zgemvt_offset_ntcol, ept, zgemvt_offset_width, 1><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj, offset_r_, offset_c_); break;
			case 2: gemvt_offset<hipDoubleComplex, zgemvt_offset_nb, zgemvt_offset_ntcol, ept, zgemvt_offset_width, 2><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj, offset_r_, offset_c_); break;
			case 3: gemvt_offset<hipDoubleComplex, zgemvt_offset_nb, zgemvt_offset_ntcol, ept, zgemvt_offset_width, 3><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj, offset_r_, offset_c_); break;
			case 4: gemvt_offset<hipDoubleComplex, zgemvt_offset_nb, zgemvt_offset_ntcol, ept, zgemvt_offset_width, 4><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj, offset_r_, offset_c_); break;
			case 5: gemvt_offset<hipDoubleComplex, zgemvt_offset_nb, zgemvt_offset_ntcol, ept, zgemvt_offset_width, 5><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj, offset_r_, offset_c_); break;
			case 6: gemvt_offset<hipDoubleComplex, zgemvt_offset_nb, zgemvt_offset_ntcol, ept, zgemvt_offset_width, 6><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj, offset_r_, offset_c_); break;
			case 7: gemvt_offset<hipDoubleComplex, zgemvt_offset_nb, zgemvt_offset_ntcol, ept, zgemvt_offset_width, 7><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj, offset_r_, offset_c_); break;
			case 8: gemvt_offset<hipDoubleComplex, zgemvt_offset_nb, zgemvt_offset_ntcol, ept, zgemvt_offset_width, 8><<<dimGrid, dimBlock, 0, stream>>>(rows_, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, threshold, conj, offset_r_, offset_c_); break;
			default: printf("irregular part %d is not supported, please extend the case statement of zgemv\n", ept_); exit(1);
		}
	}
	else
	{
		printf("ZGEMV error: Unrecognized transpose mode %c \n", trans);
		return -1;
	}

	return 0;
}

extern "C"
int kblas_zgemv2_offset(char trans, int rows, int cols,
				hipDoubleComplex alpha, hipDoubleComplex *dA, int lda,
				hipDoubleComplex *dX, int incx,
				hipDoubleComplex  beta, hipDoubleComplex *dY, int incy,
				int offset_r, int offset_c)
{
	return kblas_zgemv2_offset_driver(trans, rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, offset_r, offset_c, 0);
}

extern "C"
int kblas_zgemv2_offset_async(	char trans, int rows, int cols,
						hipDoubleComplex alpha, hipDoubleComplex *dA, int lda,
						hipDoubleComplex *dX, int incx,
						hipDoubleComplex  beta, hipDoubleComplex *dY, int incy,
						int offset_r, int offset_c,
						hipStream_t stream)
{
	return kblas_zgemv2_offset_driver(	trans, rows, cols, alpha, dA, lda, dX, incx, beta, dY, incy, offset_r, offset_c, stream);
}
