#include "hip/hip_runtime.h"
/**
 * @copyright (c) 2012- King Abdullah University of Science and
 *                      Technology (KAUST). All rights reserved.
 **/


/**
 * @file src/blas_l2/zgemv_mgpu.cu

 * KBLAS is a high performance CUDA library for subset of BLAS
 *    and LAPACK routines optimized for NVIDIA GPUs.
 * KBLAS is provided by KAUST.
 *
 * @version 3.0.0
 * @author Ahmad Abdelfattah
 * @date 2018-11-14
 **/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "gemv_mgpu_core.cuh"
#include "gemv_mgpu_offset_core.cuh"
#include "kblas_defs.h"

#if(TARGET_SM >= 30)

#define zgemvn_mgpu_bs		(32)
#define zgemvn_mgpu_ty		(4)
//#define zgemvn_mgpu_by		(2)

#define zgemvt_mgpu_bs		(32)
#define zgemvt_mgpu_ty		(4)
//#define zgemvt_mgpu_by		(2)

#else

#define zgemvn_mgpu_bs		(32)
#define zgemvn_mgpu_ty		(8)
#define zgemvn_mgpu_by		(1)

#define zgemvt_mgpu_bs		(32)
#define zgemvt_mgpu_ty		(8)
#define zgemvt_mgpu_by		(1)

#endif

extern "C"
int kblas_zscal_async(int n, hipDoubleComplex alpha, hipDoubleComplex *x, int incx, hipStream_t stream);

extern "C"
int kblas_zgemv_mgpu_driver( char trans, int rows, int cols,
						hipDoubleComplex alpha, hipDoubleComplex *dA, int lda,
						hipDoubleComplex *dX, int incx,
						hipDoubleComplex  beta, hipDoubleComplex *dY, int incy, int gpu_gid, int ngpus,
						hipStream_t stream = 0)
{
    const hipDoubleComplex z_zero = make_hipDoubleComplex(0, 0);

	if(trans == 'n' || trans == 'N')
	{
		//******** config parameters
		const int thread_x = zgemvn_mgpu_bs;
		const int thread_y = zgemvn_mgpu_ty;
		const int elements_per_thread = thread_x/(2*thread_y);
		int grid_y_n = 1 * ngpus;
		//**************************

		// scaling with beta
		//if(gpu_gid == 0)hipblasDscal(rows, beta, dY, incy);
		if(gpu_gid == 0)kblas_zscal_async(rows, beta, dY, incy, stream);
		else kblas_zscal_async(rows, z_zero, dY, incy, stream);

		int cols_ = zgemvn_mgpu_bs * ( (cols/zgemvn_mgpu_bs)/ngpus );
		if(gpu_gid < (cols/zgemvn_mgpu_bs)%ngpus) cols_ += zgemvn_mgpu_bs;
		if(gpu_gid == (cols/zgemvn_mgpu_bs)%ngpus) cols_ += cols%zgemvn_mgpu_bs;

		int mod_r = rows % zgemvn_mgpu_bs;
		int mod_c = cols_ % zgemvn_mgpu_bs;

		if(mod_r == 0)
		{
			if(mod_c == 0)
			{
				// special case
				int blocks = rows/zgemvn_mgpu_bs;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_n);
				if(blocks == 0) return 0;
				hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_special<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, gpu_gid, ngpus);
			}
			else
			{
				// generic case for columns only
				const int irregular_cols = mod_c % elements_per_thread;

				int blocks = rows/zgemvn_mgpu_bs;
				blocks += 1;	// dummy thread block
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_n);
				if(blocks == 0) return 0;
				switch(irregular_cols)
				{
					/**
					 * The kernel for irregular dimensions has an extra template parameter.
				 	 * This parameter must be among the values listed in the switch-case statement below.
				 	 * The possible values are in the range 0 - (elements_per_thread-1)
				 	 * Make sure these values are updated whenever you change the configuration parameters.
					**/
					case  0: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  1: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  1>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  2: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  2>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  3: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  3>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  4: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  4>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  5: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  5>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  6: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  6>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  7: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  7>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  8: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  8>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  9: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  9>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 10: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 10>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 11: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 11>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 12: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 12>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 13: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 13>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 14: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 14>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 15: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 15>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					default: printf("ZGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;

				}
			}
		}
		else	// mod_r != 0
		{
			if(mod_c == 0)
			{
				// generic case for columns only
				int blocks = (rows/zgemvn_mgpu_bs) + (mod_r != 0);
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_n);
				if(blocks == 0) return 0;
				hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus);
			}
			else
			{
				// generic case for rows and cols
				const int irregular_cols = mod_c % elements_per_thread;

				int blocks = (rows/zgemvn_mgpu_bs) + (mod_r != 0);

				//printf("gpu_gid = %d, cols_ = %d \n", gpu_gid, cols_);
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_n);
				if(blocks == 0) return 0;
				switch(irregular_cols)
				{
					/**
					 * The kernel for irregular dimensions has an extra template parameter.
				 	 * This parameter must be among the values listed in the switch-case statement below.
				 	 * The possible values are in the range 0 - (elements_per_thread-1)
				 	 * Make sure these values are updated whenever you change the configuration parameters.
					**/
					case  0: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  1: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  1>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  2: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  2>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  3: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  3>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  4: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  4>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  5: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  5>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  6: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  6>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  7: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  7>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  8: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  8>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case  9: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  9>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 10: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 10>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 11: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 11>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 12: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 12>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 13: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 13>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 14: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 14>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					case 15: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 15>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus); break;
					default: printf("ZGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;

				}
			}
		}
	}	// end of non-transpose case
	else if (trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
	{
		int conj;
		if(trans == 'c' || trans == 'C') conj = 1;
		else conj = 0;
		//************ config parameters
		const int thread_x = zgemvt_mgpu_bs;
		const int thread_y = zgemvt_mgpu_ty;
		const int elements_per_thread = thread_x/(2*thread_y);
		int grid_y_t = 1 * ngpus;
		//******************************

		// scaling with beta
		//if(gpu_gid == 0)hipblasDscal(cols, beta, dY, incy);
		if(gpu_gid == 0)kblas_zscal_async(cols, beta, dY, incy, stream);
		else kblas_zscal_async(cols, z_zero, dY, incy, stream);

		int cols_ = zgemvt_mgpu_bs * ( (cols/zgemvt_mgpu_bs)/ngpus );
		if(gpu_gid < (cols/zgemvt_mgpu_bs)%ngpus) cols_ += zgemvt_mgpu_bs;
		if(gpu_gid == (cols/zgemvt_mgpu_bs)%ngpus) cols_ += cols%zgemvt_mgpu_bs;

		int mod_r = rows % zgemvt_mgpu_bs;
		int mod_c = cols_ % zgemvt_mgpu_bs;

		if(mod_c == 0)
		{
			if(mod_r == 0)
			{
				// special case
				int blocks = cols_/zgemvt_mgpu_bs;
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_t);
				if(blocks == 0) return 0;
				hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_special<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, gpu_gid, ngpus, conj);
			}
			else
			{
				// mod_r != 0
				int blocks = cols_/zgemvt_mgpu_bs;
				blocks += 1;	// dummy thread block
				dim3 dimBlock(thread_x, thread_y);
				dim3 dimGrid(blocks, grid_y_t);
				if(blocks == 0) return 0;
				hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj);
			}
		}
		else	// mod_c != 0
		{
			const int irregular_cols = mod_c % elements_per_thread;

			int blocks = cols_/zgemvt_mgpu_bs + (mod_c != 0);
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_t);
			if(blocks == 0) return 0;

			switch(irregular_cols)
			{
				/**
				  * The kernel for irregular dimensions has an extra template parameter.
				  * This parameter must be among the values listed in the switch-case statement below.
				  * The possible values are in the range 0 - (elements_per_thread-1)
				  * Make sure these values are updated whenever you change the configuration parameters.
				**/
				case  0: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  1: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  1>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  2: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  2>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  3: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  3>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  4: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  4>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  5: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  5>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  6: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  6>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  7: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  7>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  8: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  8>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case  9: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  9>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 10: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 10>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 11: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 11>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 12: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 12>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 13: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 13>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 14: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 14>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				case 15: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 15>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, gpu_gid, ngpus, conj); break;
				default: printf("ZGEMV-T error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}
	}
	else
	{
		printf("ZGEMV error: Unrecognized transpose mode %c \n", trans);
		return -1;
	}

	return 0;
}
/*************************************************************************************/
extern "C"
int kblas_zgemv_mgpu_driver_offset( char trans, int rows, int cols,
						hipDoubleComplex alpha, hipDoubleComplex *dA, int lda,
						hipDoubleComplex *dX, int incx,
						hipDoubleComplex  beta, hipDoubleComplex *dY, int incy, int gpu_gid, int ngpus,
						int offset_r, int offset_c,
						hipStream_t stream = 0)
{
    const hipDoubleComplex z_zero = make_hipDoubleComplex(0, 0);
	if(trans == 'n' || trans == 'N')
	{

		//**** Config parameters
		const int thread_x = zgemvn_mgpu_bs;
		const int thread_y = zgemvn_mgpu_ty;
		const int elements_per_thread = thread_x/(2*thread_y);
		const int grid_y_n = 2 * ngpus;
		//*************************

		/** offset necessary calculation **/
		int offset_r_ = offset_r % zgemvn_mgpu_bs;
		int offset_c_ = offset_c % zgemvn_mgpu_bs;
		int total_blocks_skipped_r = offset_r / zgemvn_mgpu_bs;
		int total_blocks_skipped_c = offset_c / zgemvn_mgpu_bs;
		int my_skipped_blocks_r = total_blocks_skipped_r;
		int my_skipped_blocks_c = total_blocks_skipped_c/ngpus;
		if(gpu_gid < (total_blocks_skipped_c%ngpus)) my_skipped_blocks_c += 1;
		int ref_gpu = total_blocks_skipped_c%ngpus;
		int new_gpu_gid = (gpu_gid - ref_gpu + ngpus) % ngpus;
		// Advance pointers accordingly
		dA += my_skipped_blocks_c * zgemvn_mgpu_bs * lda;
		dA += my_skipped_blocks_r * zgemvn_mgpu_bs;
		dX += total_blocks_skipped_c * zgemvn_mgpu_bs * incx;
		dY += total_blocks_skipped_r * zgemvn_mgpu_bs * incy;
		rows -= total_blocks_skipped_r * zgemvn_mgpu_bs;
		cols -= total_blocks_skipped_c * zgemvn_mgpu_bs;
		/** end offset necessary calculation **/

		int nstripes = (cols/zgemvn_mgpu_bs) + ((cols%zgemvn_mgpu_bs) != 0);

		// scaling with beta
		if(gpu_gid == 0)kblas_zscal_async(rows-offset_r_, beta, dY+(offset_r_*incy), incy, stream);
		else kblas_zscal_async(rows-offset_r_, z_zero, dY+(offset_r_*incy), incy, stream);

		int cols_ = zgemvn_mgpu_bs * ( (cols/zgemvn_mgpu_bs)/ngpus );
		if(new_gpu_gid < (cols/zgemvn_mgpu_bs)%ngpus) cols_ += zgemvn_mgpu_bs;
		if(new_gpu_gid == (cols/zgemvn_mgpu_bs)%ngpus) cols_ += cols%zgemvn_mgpu_bs;

		int mod_r = rows % zgemvn_mgpu_bs;
		int mod_c = cols_ % zgemvn_mgpu_bs;

		if(mod_r == 0 && mod_c == 0)
		{
			// special case
			int blocks = rows/zgemvn_mgpu_bs;
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_n);
			if(blocks == 0) return 0;
			hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_special_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_);
		}
		else
		{
			// generic case for columns only
			const int irregular_cols = mod_c % elements_per_thread;

			int blocks = (rows/zgemvn_mgpu_bs) + (mod_r != 0);
			if(mod_r == 0)blocks += 1;	// dummy thread block, will return immediately if mod_r == 0

			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_n);
			if(blocks == 0) return 0;

			switch(irregular_cols)
			{
				/**
				 * The kernel for irregular dimensions has an extra template parameter.
				 * This parameter must be among the values listed in the switch-case statement below.
				 * The possible values are in the range 0 - (elements_per_thread-1)
				 * Make sure these values are updated whenever you change the configuration parameters.
				**/
				case  0: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  1: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  1>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  2: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  2>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  3: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  3>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  4: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  4>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  5: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  5>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  6: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  6>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  7: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  7>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  8: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  8>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case  9: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread,  9>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 10: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 10>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 11: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 11>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 12: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 12>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 13: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 13>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 14: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 14>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				case 15: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvn_mgpu_generic_offset<hipDoubleComplex, zgemvn_mgpu_bs, zgemvn_mgpu_bs, zgemvn_mgpu_ty, elements_per_thread, 15>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_); break;
				default: printf("ZGEMV-N error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}
	}	// end of non-transpose case
	else if (trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
	{
		int conj;
		if(trans == 'c' || trans == 'C') conj = 1;
		else conj = 0;
		//**** Config parameters
		const int thread_x = zgemvt_mgpu_bs;
		const int thread_y = zgemvt_mgpu_ty;
		const int elements_per_thread = thread_x/(2*thread_y);
		const int grid_y_t = 2 * ngpus;
		//*************************

		/** offset necessary calculation **/
		int offset_r_ = offset_r % zgemvt_mgpu_bs;
		int offset_c_ = offset_c % zgemvt_mgpu_bs;
		int total_blocks_skipped_r = offset_r / zgemvt_mgpu_bs;
		int total_blocks_skipped_c = offset_c / zgemvt_mgpu_bs;
		int my_skipped_blocks_r = total_blocks_skipped_r;
		int my_skipped_blocks_c = total_blocks_skipped_c/ngpus;
		if(gpu_gid < (total_blocks_skipped_c%ngpus)) my_skipped_blocks_c += 1;
		int ref_gpu = total_blocks_skipped_c%ngpus;
		int new_gpu_gid = (gpu_gid - ref_gpu + ngpus) % ngpus;

		//if(new_gpu_gid != 3){return 0;}
		// Advance pointers accordingly
		dA += my_skipped_blocks_c * zgemvt_mgpu_bs * lda;
		dA += my_skipped_blocks_r * zgemvt_mgpu_bs;
		dX += total_blocks_skipped_r * zgemvt_mgpu_bs * incx;
		dY += total_blocks_skipped_c * zgemvt_mgpu_bs * incy;
		rows -= total_blocks_skipped_r * zgemvt_mgpu_bs;
		cols -= total_blocks_skipped_c * zgemvt_mgpu_bs;
		/** end offset necessary calculation **/

		int nstripes = (cols/zgemvt_mgpu_bs) + ((cols%zgemvt_mgpu_bs) != 0);

		// scaling with beta
		//if(gpu_gid == 0)hipblasDscal(cols-offset_, beta, dY+(offset_*incy), incy);
		if(gpu_gid == 0)kblas_zscal_async(cols-offset_c_, beta, dY+(offset_c_*incy), incy, stream);
		else kblas_zscal_async(cols-offset_r_, z_zero, dY+(offset_r_*incy), incy, stream);

		int cols_ = zgemvt_mgpu_bs * ( (cols/zgemvt_mgpu_bs)/ngpus );
		if(new_gpu_gid < (cols/zgemvt_mgpu_bs)%ngpus) cols_ += zgemvt_mgpu_bs;
		if(new_gpu_gid == (cols/zgemvt_mgpu_bs)%ngpus) cols_ += cols%zgemvt_mgpu_bs;

		int mod_r = rows % zgemvt_mgpu_bs;
		int mod_c = cols_ % zgemvt_mgpu_bs;

		if(mod_r == 0 && mod_c == 0)
		{
			int blocks = cols_/zgemvt_mgpu_bs;
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_t);
			if(blocks == 0) return 0;
			hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_special_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj);

		}
		else
		{
			const int irregular_cols = mod_c % elements_per_thread;
			int blocks = cols_/zgemvt_mgpu_bs + (mod_c != 0);
			int gpu_last = (nstripes+ngpus-1)%ngpus;
			if(mod_c == 0 && new_gpu_gid == gpu_last) blocks += 1; // dummy thread block, will return if mod_c == 0
			dim3 dimBlock(thread_x, thread_y);
			dim3 dimGrid(blocks, grid_y_t);
			if(blocks == 0) return 0;

			switch(irregular_cols)
			{
				/**
				  * The kernel for irregular dimensions has an extra template parameter.
				  * This parameter must be among the values listed in the switch-case statement below.
				  * The possible values are in the range 0 - (elements_per_thread-1)
				  * Make sure these values are updated whenever you change the configuration parameters.
				**/
				case  0: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  0>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  1: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  1>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  2: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  2>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  3: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  3>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  4: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  4>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  5: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  5>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  6: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  6>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  7: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  7>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  8: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  8>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case  9: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread,  9>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 10: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 10>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 11: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 11>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 12: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 12>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 13: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 13>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 14: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 14>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				case 15: hipLaunchKernelGGL(HIP_KERNEL_NAME(gemvt_mgpu_generic_offset<hipDoubleComplex, zgemvt_mgpu_bs, thread_x, thread_y, elements_per_thread, 15>), dim3(dimGrid), dim3(dimBlock), 0, stream, rows, cols_, alpha, dA, lda, dX, incx, beta, dY, incy, mod_r, mod_c, new_gpu_gid, ngpus, nstripes, offset_r_, offset_c_, conj); break;
				default: printf("ZGEMV-T error: improper template parameter. Please read the inline documentation for this function. \n"); return -1;
			}
		}
	}
	else
	{
		printf("ZGEMV error: Unrecognized transpose mode %c \n", trans);
		return -1;
	}

	return 0;
}
/***********************************************************************************/
extern "C"
int kblas_zgemv_mgpu( char trans, int rows, int cols,
						hipDoubleComplex alpha, hipDoubleComplex **dA, int lda,
						hipDoubleComplex **dX, int incx,
						hipDoubleComplex  beta, hipDoubleComplex **dY, int incy,
						int ngpus,
						int offset_r, int offset_c)
{
    const int ngpus_local = ngpus;
	if(offset_r == 0 && offset_c == 0)
	{
		for(int i = 0; i < ngpus_local; i++)
		{
			hipSetDevice(gpu_lid[i]);
			kblas_zgemv_mgpu_driver(trans, rows, cols, alpha, dA[i], lda, dX[i], incx, beta, dY[i], incy, gpu_gid[i], ngpus);
		}
	}
	else
	{
		for(int i = 0; i < ngpus_local; i++)
		{
			hipSetDevice(gpu_lid[i]);
			kblas_zgemv_mgpu_driver_offset(trans, rows, cols, alpha, dA[i], lda, dX[i], incx, beta, dY[i], incy, gpu_gid[i], ngpus, offset_r, offset_c);
		}
	}

	// wait for gpus to finish
	for(int i = 0; i < ngpus_local; i++)
	{
		hipSetDevice(gpu_lid[i]);
		hipDeviceSynchronize();
	}
	return 0;
}

/*************************************************************************************/
extern "C"
int kblas_zgemv_mgpu_async( char trans, int rows, int cols,
							hipDoubleComplex alpha, hipDoubleComplex **dA, int lda,
							hipDoubleComplex **dX, int incx,
							hipDoubleComplex  beta, hipDoubleComplex **dY, int incy,
							int ngpus,
							int offset_r, int offset_c,
							hipStream_t stream[MAX_NGPUS][MAX_STREAMS])
{
    const int ngpus_local = ngpus;
	if(offset_r == 0 && offset_c == 0)
	{
		for(int i = 0; i < ngpus_local; i++)
		{
			hipSetDevice(gpu_lid[i]);
			kblas_zgemv_mgpu_driver(trans, rows, cols, alpha, dA[i], lda, dX[i], incx, beta, dY[i], incy, gpu_gid[i], ngpus, stream[i][0]);
		}
	}
	else
	{
		for(int i = 0; i < ngpus_local; i++)
		{
			hipSetDevice(gpu_lid[i]);
			kblas_zgemv_mgpu_driver_offset(trans, rows, cols, alpha, dA[i], lda, dX[i], incx, beta, dY[i], incy, gpu_gid[i], ngpus, offset_r, offset_c);
		}
	}

	return 0;
}
/*************************************************************************************/

extern "C"
int get_zgemv_mgpu_bs(char trans)
{
    if(trans == 'n' || trans == 'N')
        return zgemvn_mgpu_bs;
    else if (trans == 't' || trans == 'T' || trans == 'c' || trans == 'C')
        return zgemvt_mgpu_bs;
    else
        {printf("Error ..  input %c is not supported for gemv \n", trans); return -1;}
}
